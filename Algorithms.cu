#include "hip/hip_runtime.h"
#include "Algorithms.h"
#include "Algorithms.cuh"

#include "ExpManager.h"
#include "ThreefryGPU.h"
#include "GPUDna.cuh"

#include <cstdint>
#include <stdio.h>
#include <unistd.h>

#include <iostream>

#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>

using namespace std;

#define DEBUG 1
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}


constexpr int32_t PROMOTER_ARRAY_SIZE = 10000;

/**
 * Function to transfer data from CPU to GPU
 *
 * @param exp_m
 * @param first_gen
 */
void transfer_in(ExpManager* exp_m, bool first_gen) {
    exp_m->rng_->initDevice();
    checkCuda(hipMalloc((void**) &gpu_counters,
                         exp_m->rng_->counters().size() *
                         sizeof(unsigned long long)));

    checkCuda(hipMemcpy(gpu_counters, exp_m->rng_->counters().data(),
                         exp_m->rng_->counters().size() *
                         sizeof(unsigned long long), hipMemcpyHostToDevice));

    // TO COMPLETE
}


__device__ int32_t Threefry::Device::roulette_random(double* probs, int32_t nb_elts)
{
    double pick_one = 0.0;

    while (pick_one == 0.0)
    {
        pick_one = randomDouble();
    }

    int32_t found_org = 0;

    pick_one -= probs[0];
    while (pick_one > 0)
    {
        assert(found_org<nb_elts-1);

        pick_one -= probs[++found_org];
    }
    return found_org;
}


__constant__ double cof[6] = {  76.18009172947146,
                                -86.50532032941677,
                                24.01409824083091,
                                -1.231739572450155,
                                0.1208650973866179e-2,
                                -0.5395239384953e-5 };



// Returns the value ln[gamma(X)] for X.
// The gamma function is defined by the integral  gamma(z) = int(0, +inf, t^(z-1).e^(-t)dt).
// When the argument z is an integer, the gamma function is just the familiar factorial
// function, but offset by one, n! = gamma(n + 1).
__device__ static double gammln(double X)
{
    double x, y, tmp, ser;

    y = x = X;
    tmp = x + 5.5;
    tmp -= (x+0.5) * log(tmp);
    ser = 1.000000000190015;

    for (int8_t j = 0 ; j <= 5 ; j++)
    {
        ser += cof[j] / ++y;
    }

    return -tmp + log(2.5066282746310005 * ser / x);
}


__device__ 
int32_t Threefry::Device::binomial_random(int32_t nb_drawings, double prob)
{
    int32_t nb_success;

    // The binomial distribution is invariant under changing
    // ProbSuccess to 1-ProbSuccess, if we also change the answer to
    // NbTrials minus itself; we ll remember to do this below.
    double p;
    if (prob <= 0.5) p = prob;
    else p = 1.0 - prob;

    // mean of the deviate to be produced
    double mean = nb_drawings * p;


    if (nb_drawings < 25)
        // Use the direct method while NbTrials is not too large.
        // This can require up to 25 calls to the uniform random.
    {
        nb_success = 0;
        for (int32_t j = 1 ; j <= nb_drawings ; j++)
        {
            if (randomDouble() < p) nb_success++;
        }
    }
    else if (mean < 1.0)
        // If fewer than one event is expected out of 25 or more trials,
        // then the distribution is quite accurately Poisson. Use direct Poisson method.
    {
        double g = exp(-mean);
        double t = 1.0;
        int32_t j;
        for (j = 0; j <= nb_drawings ; j++)
        {
            t = t * randomDouble();
            if (t < g) break;
        }

        if (j <= nb_drawings) nb_success = j;
        else nb_success = nb_drawings;
    }

    else
        // Use the rejection method.
    {
        double en     = nb_drawings;
        double oldg   = gammln(en + 1.0);
        double pc     = 1.0 - p;
        double plog   = log(p);
        double pclog  = log(pc);

        // rejection method with a Lorentzian comparison function.
        double sq = sqrt(2.0 * mean * pc);
        double angle, y, em, t;
        do
        {
            do
            {
                angle = M_PI * randomDouble();
                y = tan(angle);
                em = sq*y + mean;
            } while (em < 0.0 || em >= (en + 1.0)); // Reject.

            em = floor(em); // Trick for integer-valued distribution.
            t = 1.2 * sq * (1.0 + y*y)
                * exp(oldg - gammln(em + 1.0) - gammln(en - em + 1.0) + em * plog + (en - em) * pclog);

        } while (randomDouble() > t); // Reject. This happens about 1.5 times per deviate, on average.

        nb_success = (int32_t) rint(em);
    }


    // Undo the symmetry transformation.
    if (p != prob) nb_success = nb_drawings - nb_success;

    return nb_success;
}

__device__ static int mod(int a, int b)
{

    assert(b > 0);

    while (a < 0)  a += b;
    while (a >= b) a -= b;

    return a;
}

/**
 * Run a step on the GPU
 * @param nb_indiv
 * @param w_max
 * @param selection_pressure
 * @param grid_width
 * @param grid_height
 * @param mutation_rate
 */
void run_a_step_on_GPU(int nb_indiv, double w_max, double selection_pressure, int grid_width, int grid_height, double mutation_rate) {

}

/**
 * Reallocate some data structures if needed
 * @param nb_indiv
 */
void allocate_next_gen(int nb_indiv) {

}

/**
PRNG usage:
 * For selection
        Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::REPROD,nb_indiv);
        int found_org = rng.roulette_random(probs, NEIGHBORHOOD_SIZE);
 * For mutation:
      Threefry::Device rng(gpu_counters,indiv_id,Threefry::Phase::MUTATION,nb_indivs);
      rng.binomial_random(prev_gen_size, mutation_r);
      rng.random( number );
 **/